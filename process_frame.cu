#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void invert_colors(float* input, float* output, int height, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = height * width;
    if (idx < total) {
        output[idx] = 1 - input[idx];  // Simple invert
    }
}

extern "C"
void process_frame(float* input, float* output, int height, int width) {
    int total = height * width;
    float* d_in;
    float* d_out;
    printf("height=%d\n", height);
    printf("width=%d\n", width);
    printf("total=%d\n", total);

    hipMalloc(&d_in, total * sizeof(float));
    hipMalloc(&d_out, total * sizeof(float));

    hipMemcpy(d_in, input, total*sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (total + threads - 1) / threads;
    invert_colors<<<blocks, threads>>>(d_in, d_out, height, width);
    hipDeviceSynchronize();

    hipMemcpy(output, d_out, total*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}

