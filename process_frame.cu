#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void invert_colors(unsigned char* input, unsigned char* output, int height, int width, int channels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = height * width * channels;
    if (idx < total) {
        output[idx] = 255 - input[idx];  // Simple invert
    }
}

extern "C"
void process_frame(unsigned char* input, unsigned char* output, int height, int width, int channels) {
    int total = height * width * channels;
    unsigned char* d_in;
    unsigned char* d_out;

    hipMalloc((void**)&d_in, total);
    hipMalloc((void**)&d_out, total);

    hipMemcpy(d_in, input, total, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (total + threads - 1) / threads;
    invert_colors<<<blocks, threads>>>(d_in, d_out, height, width, channels);
    hipDeviceSynchronize();

    hipMemcpy(output, d_out, total, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}

